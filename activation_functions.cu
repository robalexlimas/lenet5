#include "hip/hip_runtime.h"
#include "activation_functions.h"

__device__
float relu(float a) {
    return a < 0.0f ? 0.0f : a;
}

__device__
float tanh(float a) {
    return (2 / (1 + exp(-2 * a))) - 1;
}

__device__
float sigmoid(float a) {
    return 1 / (1 + exp(-a));
}

__device__
int activation_function(int activation) {
    int function = 0;
    switch(activation) {
        case RELU:
            function = &relu;
            break;
        case TANH:
            function = &tanh;
            break;
        case SIGMOID:
            function = &sigmoid;
            break;
    }
    return function;
}
