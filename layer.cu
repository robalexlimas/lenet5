#include "hip/hip_runtime.h"
#include "layer.h"

__global__
void conv2d(float *input, float *output, float *filters, float *bias, int width, int height, int filters_total, int shape_filters, int activation_name) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x > width || y > height || z > filters_total) return;

    float conv = 0.0f;
    for (int i=0;i<shape_filters;i++) {
        for (int j=0;j<shape_filters;j++) {
            conv += input[(z * shape_filters * shape_filters) + (y + i) * width + (x + j)] * filters[(z * shape_filters * shape_filters) + (y + i) * width + (x + j)];
        }
    }
    conv += bias[z];
    void (*activation)(int) = activation_function(activation_name);
    float result = (*activation)(conv);
    output[z * shape_filters * shape_filters + y * shape_filters + x] = result;
    __syncthreads();
}

__global__
void averagePooling2d(float *input, float *output, int width, int height, int shape_filter) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    int new_width = ceil(width / shape_filter);
    int new_height = ceil(height / shape_filter);
    if (x > new_width || y > new_height) return;

    float result = 0.0f;
    for (int i=0;i<shape_filter;i++) {
        for (int j=0;j<shape_filter;j++) {
            result += input[(y + i) * width + (x + j)];
        }
    }
    output[y * new_width + x] = result / (shape_filter * shape_filter);
    __syncthreads();
}

__global__
void dense(float *input, float *output, float *weights, float *bias, int activation_name, int units) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x > units || z > units) return;

    float result = input[x] * weights[x] + bias[x];
    void (*activation)(int) = activation_function(activation_name);
    output[x] = (*activation)(result);
    __syncthreads();
}
