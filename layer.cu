#include "hip/hip_runtime.h"
#include "layer.h"

__global__
void conv2d(float *input, float *output, float *filters, int widht, int height, int filters, int shape_filters, int activation_name) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    void (*activation)(int) = activation_function(activation_name);
    float result = (*activation)(0.0f);
}

__global__
void averagePooling2d(float *input, float *output, int shape_filter) {

}

__global__
void dense(float *input, float *output, int activation_name) {
    void (*activation)(int) = activation_function(activation_name);
    float result = (*activation)(0.0f);
}
